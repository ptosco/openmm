#include "hip/hip_runtime.h"
float2 bondParams = PARAMS[index];
real deltaIdeal = r-bondParams.x;
real deltaIdeal2 = deltaIdeal*deltaIdeal;
energy += 0.5f*MMFF_BOND_C1*bondParams.y*deltaIdeal2*(1.0f + CUBIC_K*deltaIdeal + QUARTIC_K*deltaIdeal2);
real dEdR = MMFF_BOND_C1*bondParams.y*deltaIdeal*(1.0f + 1.5f*CUBIC_K*deltaIdeal + 2.0f*QUARTIC_K*deltaIdeal2);
