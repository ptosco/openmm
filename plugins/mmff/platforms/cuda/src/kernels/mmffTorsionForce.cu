#include "hip/hip_runtime.h"
float3 torsionK = PARAMS[index];
const real torsionParams1 = torsionK.y + 0.5f*(torsionK.x + torsionK.z);
const real torsionParams2 = 0.5f*(3.0f*torsionK.z - torsionK.x);
const real torsionParams3 = -torsionK.y;
const real torsionParams4 = -2.0f*torsionK.z;
if (theta < 0)
    theta += PI;
else
    theta -= PI;
cosangle = -cosangle;
real cosFactor = cosangle;
real dEdAngle = -torsionParams2;
real rbEnergy = torsionParams1;
rbEnergy += torsionParams2*cosFactor;
dEdAngle -= 2.0f*torsionParams3*cosFactor;
cosFactor *= cosangle;
dEdAngle -= 3.0f*torsionParams4*cosFactor;
rbEnergy += torsionParams3*cosFactor;
cosFactor *= cosangle;
rbEnergy += torsionParams4*cosFactor;
energy += rbEnergy;
dEdAngle *= SIN(theta);
