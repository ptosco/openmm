#include "hip/hip_runtime.h"
{
    const real eleBuf = 0.005f;
    real rBuf = r + eleBuf;
    real rBufInv = RECIP(rBuf);
#if USE_EWALD
    bool needCorrection = hasExclusions && isExcluded && atom1 != atom2 && atom1 < NUM_ATOMS && atom2 < NUM_ATOMS;
    bool includeInteraction = ((!isExcluded && r2 < CUTOFF_SQUARED) || needCorrection);
    const real alphaR = EWALD_ALPHA*rBuf;
    const real expAlphaRSqr = EXP(-alphaR*alphaR);
    const real prefactor = 138.935456f*posq1.w*posq2.w*rBufInv;

#ifdef USE_DOUBLE_PRECISION
    const real erfcAlphaR = erfc(alphaR);
#else
    // This approximation for erfc is from Abramowitz and Stegun (1964) p. 299.  They cite the following as
    // the original source: C. Hastings, Jr., Approximations for Digital Computers (1955).  It has a maximum
    // error of 1.5e-7.

    const real t = RECIP(1.0f+0.3275911f*alphaR);
    const real erfcAlphaR = (0.254829592f+(-0.284496736f+(1.421413741f+(-1.453152027f+1.061405429f*t)*t)*t)*t)*t*expAlphaRSqr;
#endif
    real tempForce = 0.0f;
    if (needCorrection) {
        // Subtract off the part of this interaction that was included in the reciprocal space contribution.

        if (1-erfcAlphaR > 1e-6) {
            real erfAlphaR = ERF(alphaR); // Our erfc approximation is not accurate enough when r is very small, which happens with Drude particles.
            tempForce = -prefactor*(erfAlphaR-alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
            tempEnergy += -prefactor*erfAlphaR;
        }
        else {
            includeInteraction = false;
            tempEnergy -= TWO_OVER_SQRT_PI*EWALD_ALPHA*138.935456f*posq1.w*posq2.w;
        }
    }
    else {
#if HAS_VDW
    if (includeInteraction) {
        bool haveDonor = false;
        bool haveDAPair = (params1.y < 0.0f && params2.z < 0.0f)
            || (params1.z < 0.0f && params2.y < 0.0f);
        real G_t_alpha1 = params1.y;
        if (params1.y < 0.0f) {
            haveDonor = true;
            G_t_alpha1 = -G_t_alpha1;
        }
        real G_t_alpha2 = params2.y;
        if (params2.y < 0.0f) {
            haveDonor = true;
            G_t_alpha2 = -G_t_alpha2;
        }
        const real B = 0.2f;
        const real Beta = 12.0f;
        const real c = 7.5797344e-4f;
        const real DARAD = 0.8f;
        const real DAEPS = 0.5f;
        real gamma = (params1.x - params2.x) / (params1.x + params2.x);
        real sigma = 0.5f * (params1.x + params2.x) * (1.0f + (haveDonor
                     ? 0.0f : B * (1.0f - exp(-Beta * gamma * gamma))));
        real alpha_d_N1 = ((params1.z < 0.0f) ? -params1.z : params1.z);
        real alpha_d_N2 = ((params2.z < 0.0f) ? -params2.z : params2.z);
        real sigmaSq = sigma * sigma;
        real epsilon = c * G_t_alpha1 * G_t_alpha2
                       / ((sqrt(alpha_d_N1) + sqrt(alpha_d_N2)) * sigmaSq * sigmaSq * sigmaSq);
        if (haveDAPair) {
            sigma *= DARAD;
            epsilon *= DAEPS;
        }
        real r6 = r2*r2*r2;
        real r7 = r6*r;
        real sigma7 = sigma*sigma;
        sigma7 = sigma7*sigma7*sigma7*sigma;
        real rho = r7 + sigma7*0.12f;
        real invRho = RECIP(rho);
        real tau = 1.07f/(r + 0.07f*sigma);
        real tau7 = tau*tau*tau;
        tau7 = tau7*tau7*tau;
        real dTau = tau/1.07f;
        real tmp = sigma7*invRho;
        real gTau = epsilon*tau7*r6*1.12f*tmp*tmp;
        real vdwEnergy = epsilon*sigma7*tau7*((sigma7*1.12f*invRho)-2.0f);
        real deltaE = -7.0f*(dTau*vdwEnergy+gTau);
    #if USE_VDW_SWITCH
        if (r > VDW_SWITCH_CUTOFF) {
            real x = r-VDW_SWITCH_CUTOFF;
            real taper = 1+x*x*x*(VDW_SWITCH_C3+x*(VDW_SWITCH_C4+x*VDW_SWITCH_C5));
            real dtaper = x*x*(3*VDW_SWITCH_C3+x*(4*VDW_SWITCH_C4+x*5*VDW_SWITCH_C5));
            deltaE = vdwEnergy*dtaper + deltaE*taper;
            vdwEnergy *= taper;
        }
    #endif
        dEdR -= deltaE*invR;
        tempEnergy += vdwEnergy + prefactor*erfcAlphaR;
    }
#else
        tempEnergy += includeInteraction ? prefactor*erfcAlphaR : 0;
#endif
        tempForce = prefactor*(erfcAlphaR+alphaR*expAlphaRSqr*TWO_OVER_SQRT_PI);
    }
    dEdR += includeInteraction ? tempForce*rBufInv*rBufInv : 0;
#else
#ifdef USE_CUTOFF
    bool includeInteraction = (!isExcluded && r2 < CUTOFF_SQUARED);
#else
    bool includeInteraction = (!isExcluded);
#endif
    real tempForce = 0.0f;
  #if HAS_VDW
    if (includeInteraction) {
        bool haveDonor = false;
        bool haveDAPair = (params1.y < 0.0f && params2.z < 0.0f)
            || (params1.z < 0.0f && params2.y < 0.0f);
        real G_t_alpha1 = params1.y;
        if (params1.y < 0.0f) {
            haveDonor = true;
            G_t_alpha1 = -G_t_alpha1;
        }
        real G_t_alpha2 = params2.y;
        if (params2.y < 0.0f) {
            haveDonor = true;
            G_t_alpha2 = -G_t_alpha2;
        }
        const real B = 0.2f;
        const real Beta = 12.0f;
        const real c = 7.5797344e-4f;
        const real DARAD = 0.8f;
        const real DAEPS = 0.5f;
        real gamma = (params1.x - params2.x) / (params1.x + params2.x);
        real sigma = 0.5f * (params1.x + params2.x) * (1.0f + (haveDonor
                     ? 0.0f : B * (1.0f - exp(-Beta * gamma * gamma))));
        real alpha_d_N1 = ((params1.z < 0.0f) ? -params1.z : params1.z);
        real alpha_d_N2 = ((params2.z < 0.0f) ? -params2.z : params2.z);
        real sigmaSq = sigma * sigma;
        real epsilon = c * G_t_alpha1 * G_t_alpha2
                       / ((sqrt(alpha_d_N1) + sqrt(alpha_d_N2)) * sigmaSq * sigmaSq * sigmaSq);
        if (haveDAPair) {
            sigma *= DARAD;
            epsilon *= DAEPS;
        }
        real r6 = r2*r2*r2;
        real r7 = r6*r;
        real sigma7 = sigma*sigma;
        sigma7 = sigma7*sigma7*sigma7*sigma;
        real rho = r7 + sigma7*0.12f;
        real invRho = RECIP(rho);
        real tau = 1.07f/(r + 0.07f*sigma);
        real tau7 = tau*tau*tau;
        tau7 = tau7*tau7*tau;
        real dTau = tau/1.07f;
        real tmp = sigma7*invRho;
        real gTau = epsilon*tau7*r6*1.12f*tmp*tmp;
        real vdwEnergy = epsilon*sigma7*tau7*((sigma7*1.12f*invRho)-2.0f);
        real deltaE = -7.0f*(dTau*vdwEnergy+gTau);
      #if USE_VDW_SWITCH
        if (r > VDW_SWITCH_CUTOFF) {
            real x = r-VDW_SWITCH_CUTOFF;
            real taper = 1+x*x*x*(VDW_SWITCH_C3+x*(VDW_SWITCH_C4+x*VDW_SWITCH_C5));
            real dtaper = x*x*(3*VDW_SWITCH_C3+x*(4*VDW_SWITCH_C4+x*5*VDW_SWITCH_C5));
            deltaE = vdwEnergy*dtaper + deltaE*taper;
            vdwEnergy *= taper;
        }
      #endif
        dEdR -= deltaE*invR;
        tempEnergy += vdwEnergy;
    }
  #endif
#if HAS_COULOMB
  #ifdef USE_CUTOFF
    const real prefactor = 138.935456f*posq1.w*posq2.w;
    tempForce += prefactor*(rBufInv - 2.0f*REACTION_FIELD_K*r2);
    tempEnergy += includeInteraction ? prefactor*(rBufInv + REACTION_FIELD_K*rBuf*rBuf - REACTION_FIELD_C) : 0;
  #else
    const real prefactor = 138.935456f*posq1.w*posq2.w*rBufInv;
    tempForce += prefactor;
    tempEnergy += includeInteraction ? prefactor : 0;
  #endif
#endif
    dEdR += includeInteraction ? tempForce*rBufInv*invR : 0;
#endif
}
