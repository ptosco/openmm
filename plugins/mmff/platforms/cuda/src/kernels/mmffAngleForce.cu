#include "hip/hip_runtime.h"
float2 angleParams = PARAMS[index];
real deltaIdeal = theta*RAD_TO_DEG-angleParams.x;
real dEdAngle = MMFF_ANGLE_C1*angleParams.y;
if (angleParams.y < 0.0f) {
    const real MIN_SINE = 1.0e-8f;
    real cosine = cos(theta);
    real sine2 = 1.0f - cosine*cosine;
    sine = (sine2 < 0.0f) ? MIN_SINE : sqrt(sine2);
    if (sine < MIN_SINE) sine = MIN_SINE;
    energy += -dEdAngle*(1.0f + cosine);
    dEdAngle *= -sine;
}
else {
    real p = CUBIC_K*deltaIdeal;
    energy += 0.5f*dEdAngle*deltaIdeal*deltaIdeal*(1.0f + p);
    dEdAngle *= RAD_TO_DEG*deltaIdeal*(1.0f + 1.5f*p);
}
