#include "hip/hip_runtime.h"
{
//#define VDWDEBUG
#ifdef USE_CUTOFF
    unsigned int includeInteraction = (!isExcluded && r2 < CUTOFF_SQUARED);
#else
    unsigned int includeInteraction = (!isExcluded);
#endif
    bool haveDonor = false;
    bool haveDAPair = (params1.y < 0.0f && params2.z < 0.0f)
        || (params1.z < 0.0f && params2.y < 0.0f);
    real G_t_alpha1 = params1.y;
    if (params1.y < 0.0f) {
        haveDonor = true;
        G_t_alpha1 = -G_t_alpha1;
    }
    real G_t_alpha2 = params2.y;
    if (params2.y < 0.0f) {
        haveDonor = true;
        G_t_alpha2 = -G_t_alpha2;
    }
    const real B = 0.2f;
    const real Beta = 12.0f;
    const real C4 = 7.5797344e-4f;
    const real DARAD = 0.8f;
    const real DAEPS = 0.5f;
    real gamma = (params1.x - params2.x) / (params1.x + params2.x);
    real sigma = 0.5f * (params1.x + params2.x) * (1.0f + (haveDonor
                 ? 0.0f : B * (1.0f - exp(-Beta * gamma * gamma))));
    real alpha_d_N1 = ((params1.z < 0.0f) ? -params1.z : params1.z);
    real alpha_d_N2 = ((params2.z < 0.0f) ? -params2.z : params2.z);
    real sigmaSq = sigma * sigma;
    real epsilon = C4 * G_t_alpha1 * G_t_alpha2
                   / ((sqrt(alpha_d_N1) + sqrt(alpha_d_N2)) * sigmaSq * sigmaSq * sigmaSq);
#ifdef VDWDEBUG
    bool sc = false;
#endif
    if (haveDAPair) {
#ifdef VDWDEBUG
        sc = true;
#endif
        sigma *= DARAD;
        epsilon *= DAEPS;
    }
#ifdef VDWDEBUG
    printf("GIaI=%f,GJaJ=%f,aI/NI=%f,aJ/NJ=%f,s1=%f,s2=%f,s=%f,e=%f,sc=%d\n",G_t_alpha1,G_t_alpha2,alpha_d_N1,alpha_d_N2,params1.x,params2.x,sigma,epsilon,sc);
#endif
    real r6 = r2*r2*r2;
    real r7 = r6*r;
    real sigma7 = sigma*sigma;
    sigma7 = sigma7*sigma7*sigma7*sigma;
    real rho = r7 + sigma7*0.12f;
    real invRho = RECIP(rho);
    real tau = 1.07f/(r + 0.07f*sigma);
    real tau7 = tau*tau*tau;
    tau7 = tau7*tau7*tau;
    real dTau = tau/1.07f;
    real tmp = sigma7*invRho;
    real gTau = epsilon*tau7*r6*1.12f*tmp*tmp;
    real termEnergy = epsilon*sigma7*tau7*((sigma7*1.12f*invRho)-2.0f);
    real deltaE = -7.0f*(dTau*termEnergy+gTau);
#ifdef USE_CUTOFF
    if (r > TAPER_CUTOFF) {
        real x = r-TAPER_CUTOFF;
        real taper = 1+x*x*x*(TAPER_C3+x*(TAPER_C4+x*TAPER_C5));
        real dtaper = x*x*(3*TAPER_C3+x*(4*TAPER_C4+x*5*TAPER_C5));
        deltaE = termEnergy*dtaper + deltaE*taper;
        termEnergy *= taper;
    }
#endif
    tempEnergy += (includeInteraction ? termEnergy : 0);
    dEdR -= (includeInteraction ? deltaE*invR : 0);
}
